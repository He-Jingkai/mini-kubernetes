#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <sys/time.h>
#include <stdio.h>
#include <math.h>

const int Row=1024;
const int Col=1024;

__global__
void matrix_mul_gpu(int *M, int* N, int* P, int width)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    int sum = 0;
    for(int k=0;k<width;k++){
        int a = M[j*width+k];
        int b = N[k*width+i];
        sum += a*b;
    }
    P[j*width+i] = sum;
}

void printMatrix(int* mat, int nBytes){
  printf("matrix mul cuda result is(1024*1024):\n");
  for(int i=0; i < nBytes; i++){
    printf("%d ", mat[i]);
    if(i!=0 && i%1023 == 0)
      printf("\n");
  }
}

int main()
{
    struct timeval start, end;
    gettimeofday( &start, NULL );

    int *A = (int *)malloc(sizeof(int) * Row * Col);
    int *B = (int *)malloc(sizeof(int) * Row * Col);
    int *C = (int *)malloc(sizeof(int) * Row * Col);
    //malloc device memory
    int *d_dataA, *d_dataB, *d_dataC;
    hipMalloc((void**)&d_dataA, sizeof(int) *Row*Col);
    hipMalloc((void**)&d_dataB, sizeof(int) *Row*Col);
    hipMalloc((void**)&d_dataC, sizeof(int) *Row*Col);
    //set value
    for (int i = 0; i < Row*Col; i++) {
        A[i] = 90;
        B[i] = 10;
    }

    hipMemcpy(d_dataA, A, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
    hipMemcpy(d_dataB, B, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
    dim3 threadPerBlock(16, 16);
    dim3 blockNumber((Col+threadPerBlock.x-1)/ threadPerBlock.x, (Row+threadPerBlock.y-1)/ threadPerBlock.y );

    matrix_mul_gpu << <blockNumber, threadPerBlock >> > (d_dataA, d_dataB, d_dataC, Col);
    //拷贝计算数据-一级数据指针
    hipMemcpy(C, d_dataC, sizeof(int) * Row * Col, hipMemcpyDeviceToHost);

    printMatrix(C, Row * Col);
    //释放内存
    free(A);
    free(B);
    free(C);
    hipFree(d_dataA);
    hipFree(d_dataB);
    hipFree(d_dataC);

    gettimeofday( &end, NULL );
    int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    printf("total time is %d ms\n", timeuse/1000);

    return 0;
}
