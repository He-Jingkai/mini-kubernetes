#include <stdio.h>
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

using namespace std;

#define CHECK(call)                                                    \
  {                                                                    \
    const hipError_t error = call;                                    \
    if (error != hipSuccess) {                                        \
      printf("ERROR: %s:%d,", __FILE__, __LINE__);                     \
      printf("code:%d,reason:%s\n", error, hipGetErrorString(error)); \
      exit(1);                                                         \
    }                                                                  \
  }

void initDevice(int devNum) {
  int dev = devNum;
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  printf("Using device %d: %s\n", dev, deviceProp.name);
  CHECK(hipSetDevice(dev));
}

void initialData(float* ip, int size) {
  time_t t;
  srand((unsigned)time(&t));
  for (int i = 0; i < size; i++)
    ip[i] = (float)(rand() & 0xffff) / 1000.0f;
}

void printMatrix(float* mat, int nBytes){
  printf("matrix add cuda result is(4096*4096):\n");
  for(int i=0; i < nBytes; i++){
    printf("%f ", mat[i]);
    if(i!=0 && i%4095 == 0)
      printf("\n");
  }
}

__global__ void sumMatrix(float* MatA, float* MatB, float* MatC, int nx,
                          int ny) {
  int ix = threadIdx.x + blockDim.x * blockIdx.x;
  int iy = threadIdx.y + blockDim.y * blockIdx.y;
  int idx = ix + iy * ny;
  if (ix < nx && iy < ny)
    MatC[idx] = MatA[idx] + MatB[idx];
}

int main(int argc, char** argv) {
  //init dev
  initDevice(0);

  int nx = 1 << 12;
  int ny = 1 << 12;
  int nBytes = nx * ny * sizeof(float);

  float* A_host = (float*)malloc(nBytes);
  float* B_host = (float*)malloc(nBytes);
  float* C_from_gpu = (float*)malloc(nBytes);
  initialData(A_host, nx * ny);
  initialData(B_host, nx * ny);

  float* A_dev = NULL;
  float* B_dev = NULL;
  float* C_dev = NULL;
  CHECK(hipMalloc((void**)&A_dev, nBytes));
  CHECK(hipMalloc((void**)&B_dev, nBytes));
  CHECK(hipMalloc((void**)&C_dev, nBytes));

  CHECK(hipMemcpy(A_dev, A_host, nBytes, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(B_dev, B_host, nBytes, hipMemcpyHostToDevice));

  dim3 threadsPerBlock(32, 32);

  dim3 numBlocks((nx - 1) / threadsPerBlock.x + 1,
                 (ny - 1) / threadsPerBlock.y + 1);

  sumMatrix<<<numBlocks, threadsPerBlock>>>(A_dev, B_dev, C_dev, nx, ny);
  CHECK(hipDeviceSynchronize());

  CHECK(hipMemcpy(C_from_gpu, C_dev, nBytes, hipMemcpyDeviceToHost));
  printMatrix(C_from_gpu, nx * ny);
  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);
  free(A_host);
  free(B_host);
  free(C_from_gpu);
  hipDeviceReset();

  return 0;
}
